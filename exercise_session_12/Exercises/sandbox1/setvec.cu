#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCKSIZE 256

__global__ void kern_set_val (float *gpu_ptr, float value, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  //TO DO: evaluate the value of i
  gpu_ptr[i] = value;
}

int main () {
  int i, failed=0;
  int N = 1024;   // size of vector
  float *ptr;     // Host pointer
  float *gpu_ptr; // Device pointer

  /* Allocate vector in Host*/
  ptr = (float *)malloc(sizeof(float)*N);
  /* Allocate vector in Device*/
  hipMalloc (&gpu_ptr, sizeof(float)*N);

  //TO DO : write kernel invocation here
  kern_set_val<<<N/BLOCKSIZE,BLOCKSIZE>>>(gpu_ptr, 11,11);

  hipDeviceSynchronize ();

  //TO DO : copy data to host DONE
  hipMemcpy(ptr, gpu_ptr, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipFree (gpu_ptr);

  /* Now check that it did what we want */

  for (i = 0; i < 10; i++)//first ten values are written
    printf ("%f\t", ptr[i]);
  printf ("\n");
  for (i = N-10; i < N; i++)//last ten values are written
    printf ("%f\t", ptr[i]);
  printf ("\n");

  for (i = 0; i < N; i++) {//All values are compared
    if (fabs(ptr[i]-11.0) > 1e-8) {
      failed=1;
    }
  }
  if (failed) {
    printf ("FAILED !!\n");
  } else {
    printf ("PASSED !!\n");
  }
  free (ptr);
}
